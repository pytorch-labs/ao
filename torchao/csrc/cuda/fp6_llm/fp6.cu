#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdexcept>
#include <cstring>

// reference implementation. this doesn't have a lot of bit manipulation, so it's less error-prone
// this is not exposed to PyTorch
__device__ __host__ static uint8_t fp32_to_fp6_ref(float a) {
#ifndef __CUDA_ARCH__
    if (std::isnan(a) | std::isinf(a))
        throw std::invalid_argument("Encounter +/-inf or NaN, which is not representable in FP6.");
    if (std::abs(a) >= 30.0f)
        throw std::invalid_argument("FP6 overflow. FP6 cannot represent +/-inf.");
#endif

    a *= 0x1p-124;  // 2^(127-3)
    uint32_t bits;
    std::memcpy(&bits, &a, sizeof(a));

    uint8_t sign = bits >> 31u << 5u;
    uint8_t exp_and_man = (bits >> 21u) & 0x1Fu;
    uint8_t result = sign | exp_and_man;

    // round to nearest even
    uint32_t remainder = bits << 11u;
    if ((remainder > 0x8000'0000u) || ((remainder == 0x8000'0000u) && (result & 1u))) {
        result += 1;
    }

    return result;
}

// we need to do this because C++17 does not allow using struct as template non-type parameter
// use the upper 16 bits for num exponent, lower 16 bits for num mantissa
static constexpr uint32_t encode_fp_spec(uint32_t n_exp_bits, uint32_t n_man_bits) {
    return (n_exp_bits << 16u) | n_man_bits;
}

static constexpr uint32_t FP32_SPEC = encode_fp_spec(8u, 23u);
static constexpr uint32_t FP16_SPEC = encode_fp_spec(5u, 10u);
static constexpr uint32_t BF16_SPEC = encode_fp_spec(8u, 7u);

// NOTE: only works for len < 32
__device__ __host__ static constexpr uint32_t ones_mask(uint32_t len) { return (1u << len) - 1u; }

// inspired by __internal_float2half() and float2half() from "cuda_fp16.hpp"
template <typename T, uint32_t FP_SPEC>
__device__ __host__ static uint8_t bits_to_fp6(T bits) {
    constexpr uint32_t N_EXP = FP_SPEC >> 16u;
    constexpr uint32_t N_MAN = FP_SPEC & ones_mask(16u);
    constexpr uint32_t N_EXP_MAN = N_EXP + N_MAN;

    // sanity checks. will be removed in template specialization.
#ifndef __CUDA_ARCH__
    if (N_EXP < 3)
        throw std::invalid_argument("Number of exponent bits must be >= 3.");
    if (N_MAN < 3)
        throw std::invalid_argument("Number of mantissa bits must be >= 3.");
#endif

    T remainder = 0u;
    T sign = bits >> N_EXP_MAN << 5u;
    bits &= ones_mask(N_EXP_MAN);  // clear sign bit
    T result;

    constexpr uint32_t EXP_BIAS_DIFF = ones_mask(N_EXP - 1u) - 3u;

    // only checks for invalid values on CPU, since we can't throw exception in CUDA
#ifndef __CUDA_ARCH__
    // all exponent bits are 1s
    if (bits >= (ones_mask(N_EXP) << N_MAN))
        throw std::invalid_argument("Encounter +/-inf or NaN, which is not representable in FP6.");
    // max FP6 (28) + half of least significand (2) = 30 (assume N_MAN >= 3)
    if (bits >= (((EXP_BIAS_DIFF + 7u) << N_MAN) | (0x7u << (N_MAN - 3u))))
        throw std::invalid_argument("FP6 overflow. FP6 cannot represent +/-inf.");
#endif

    // FP6 normal number (E>=001)
    if (bits >= ((EXP_BIAS_DIFF + 1u) << N_MAN)) {
        remainder = bits << (1u + N_EXP + 2u);
        bits -= (EXP_BIAS_DIFF << N_MAN);  // update exponent
        result = sign | (bits >> (N_MAN - 2u));
    }
    // FP6 subnormal number (more than half of min FP6 subnormal = 0.0625 * 0.5)
    else if (bits > ((EXP_BIAS_DIFF - 2u) << N_MAN)) {
        T exp = bits >> N_MAN;
        T man = bits & ones_mask(N_MAN);

        // to make subnormal FP6 from normal FP16
        // step 1: add implicit 1 to mantissa
        man |= (1u << N_MAN);

        // step 2: shift mantissa right so that exponent value is equal to
        // exponent value of FP6 subnormal, which is -2 (equivalent to E=001)
        T shift = EXP_BIAS_DIFF + 1u - exp;
        remainder = man << (1u + N_EXP + 2u + shift);
        result = sign | (man >> (shift + (N_MAN - 2u)));  // implicit E=000
    }
    // FP6 underflow. E=000, M=00
    else {
        result = sign;
    }

    // round to nearest even
    constexpr T HALF_REMAINDER = 1u << N_EXP_MAN;
    if ((remainder > HALF_REMAINDER) || ((remainder == HALF_REMAINDER) && (result & 0x1u))) {
        result += 1;
    }
    return result;
}

template <typename T, uint32_t FP_SPEC>
__device__ __host__ static void bits_4_to_fp6_4_packed(const T *bits_ptr, uint8_t *fp6_ptr) {
    uint8_t val0 = bits_to_fp6<T, FP_SPEC>(bits_ptr[0]);
    uint8_t val1 = bits_to_fp6<T, FP_SPEC>(bits_ptr[1]);
    uint8_t val2 = bits_to_fp6<T, FP_SPEC>(bits_ptr[2]);
    uint8_t val3 = bits_to_fp6<T, FP_SPEC>(bits_ptr[3]);

    fp6_ptr[0] = (val0 << 2) | (val1 >> 4);  // 0000 0011
    fp6_ptr[1] = (val1 << 4) | (val2 >> 2);  // 1111 2222
    fp6_ptr[2] = (val2 << 6) | (val3);       // 2233 3333
}

// assume the lower 6 bits contain the data
__device__ __host__ static float fp6_to_fp32(const uint8_t a) {
    // we shift the bits so that sign, exponent, and mantissa bits are in their correct positions in FP32.
    // this also handles subnormal numbers correctly.
    // FP6:                                  SE EEMM
    // FP32: S000 00EE EMM0 0000 0000 0000 0000 0000
    uint32_t bits = a;  // bit extension
    uint32_t sign = bits >> 5u << 31u;
    uint32_t exp_and_man = (bits & 0x1Fu) << 21u;
    uint32_t result_bits = sign | exp_and_man;

    // the result will be off by the difference in exponent bias (3 in FP6 and 127 in FP32)
    // we can correct this by direct FP32 multiplication, which also handles subnormal numbers.
    float result;
    std::memcpy(&result, &result_bits, sizeof(result));
    return result * 0x1p124;  // 2^(127-3)
}

__device__ __host__ static void fp6_4_packed_to_fp32_4(const uint8_t *fp6_ptr, float *fp32_ptr) {
    uint8_t bits0 = fp6_ptr[0];  // 0000 0011
    uint8_t bits1 = fp6_ptr[1];  // 1111 2222
    uint8_t bits2 = fp6_ptr[2];  // 2233 3333

    fp32_ptr[0] = fp6_to_fp32(bits0 >> 2);
    fp32_ptr[1] = fp6_to_fp32(((bits0 & 0x3u) << 4) | (bits1 >> 4));
    fp32_ptr[2] = fp6_to_fp32(((bits1 & 0xFu) << 2) | (bits2 >> 6));
    fp32_ptr[3] = fp6_to_fp32(bits2 & 0x3Fu);
}

__global__ void fp6_packed_to_fp32_kernel(const uint8_t *fp6_ptr, float *fp32_ptr, int n) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
    if (idx < n)
        fp6_4_packed_to_fp32_4(fp6_ptr + idx, fp32_ptr + idx / 3 * 4);
}

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <torch/library.h>

namespace torchao {

// this is useful for debugging
at::Tensor to_fp6_unpacked_cpu(at::Tensor fp_tensor) {
    TORCH_CHECK(fp_tensor.is_contiguous());
    TORCH_CHECK(fp_tensor.is_cpu());

    at::TensorOptions options = at::TensorOptions().dtype(torch::kUInt8).device(fp_tensor.device());
    at::Tensor fp6_tensor = at::empty(fp_tensor.sizes(), options);
    uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();

    int n = fp_tensor.numel();
    auto dtype = fp_tensor.dtype();

    if (dtype == torch::kFloat32) {
        const uint32_t *fp32_ptr = reinterpret_cast<uint32_t *>(fp_tensor.data_ptr<float>());

        #pragma omp parallel for
        for (int i = 0; i < n; i++)
            fp6_ptr[i] = bits_to_fp6<uint32_t, FP32_SPEC>(fp32_ptr[i]);

    } else if (dtype == torch::kFloat16) {
        const uint16_t *fp16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::Half>());

        #pragma omp parallel for
        for (int i = 0; i < n; i++)
            fp6_ptr[i] = bits_to_fp6<uint16_t, FP16_SPEC>(fp16_ptr[i]);

    } else if (dtype == torch::kBFloat16) {
        const uint16_t *bf16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::BFloat16>());

        #pragma omp parallel for
        for (int i = 0; i < n; i++)
            fp6_ptr[i] = bits_to_fp6<uint16_t, BF16_SPEC>(bf16_ptr[i]);

    } else {
        throw std::invalid_argument("Only FP32, FP16, and BF16 inputs are accepted.");
    }

    return fp6_tensor;
}

template <typename T, uint32_t FP_SPEC>
__global__ void bits_to_fp6_unpacked_kernel(const T *bits_ptr, uint8_t *fp6_ptr, int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        fp6_ptr[idx] = bits_to_fp6<T, FP_SPEC>(bits_ptr[idx]);
}

// this is useful for debugging
at::Tensor to_fp6_unpacked_cuda(at::Tensor fp_tensor) {
    TORCH_CHECK(fp_tensor.is_contiguous());
    TORCH_CHECK(fp_tensor.is_cuda());

    at::TensorOptions options = at::TensorOptions().dtype(torch::kUInt8).device(fp_tensor.device());
    at::Tensor fp6_tensor = at::empty(fp_tensor.sizes(), options);
    uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();

    int n = fp_tensor.numel();
    auto dtype = fp_tensor.dtype();

    constexpr int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    if (dtype == torch::kFloat32) {
        const uint32_t *fp32_ptr = reinterpret_cast<uint32_t *>(fp_tensor.data_ptr<float>());
        bits_to_fp6_unpacked_kernel<uint32_t, FP32_SPEC><<<grid_size, block_size>>>(fp32_ptr, fp6_ptr, n);

    } else if (dtype == torch::kFloat16) {
        const uint16_t *fp16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::Half>());
        bits_to_fp6_unpacked_kernel<uint16_t, FP16_SPEC><<<grid_size, block_size>>>(fp16_ptr, fp6_ptr, n);

    } else if (dtype == torch::kBFloat16) {
        const uint16_t *bf16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::BFloat16>());
        bits_to_fp6_unpacked_kernel<uint16_t, BF16_SPEC><<<grid_size, block_size>>>(bf16_ptr, fp6_ptr, n);

    } else {
        throw std::invalid_argument("Only FP32, FP16, and BF16 inputs are accepted.");
    }

    return fp6_tensor;
}

at::Tensor to_fp6_packed_cpu(at::Tensor fp_tensor) {
    TORCH_CHECK(fp_tensor.is_contiguous());
    TORCH_CHECK(fp_tensor.is_cpu());
    TORCH_CHECK(fp_tensor.ndimension() == 2);

    int M = fp_tensor.size(0);
    int N = fp_tensor.size(1);
    TORCH_CHECK(N % 4 == 0, "Last dimension must be a multiple of 4, receives ", N);

    at::TensorOptions options = at::TensorOptions().dtype(torch::kUInt8).device(fp_tensor.device());
    at::Tensor fp6_tensor = at::empty({M, N * 3 / 4}, options);
    uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();

    int n = fp_tensor.numel();
    auto dtype = fp_tensor.dtype();

    if (dtype == torch::kFloat32) {
        const uint32_t *fp32_ptr = reinterpret_cast<uint32_t *>(fp_tensor.data_ptr<float>());

        #pragma omp parallel for
        for (int i = 0; i < n; i += 4)
            bits_4_to_fp6_4_packed<uint32_t, FP32_SPEC>(fp32_ptr + i, fp6_ptr + i / 4 * 3);

    } else if (dtype == torch::kFloat16) {
        const uint16_t *fp16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::Half>());

        #pragma omp parallel for
        for (int i = 0; i < n; i += 4)
            bits_4_to_fp6_4_packed<uint16_t, FP16_SPEC>(fp16_ptr + i, fp6_ptr + i / 4 * 3);

    } else if (dtype == torch::kBFloat16) {
        const uint16_t *bf16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::BFloat16>());

        #pragma omp parallel for
        for (int i = 0; i < n; i += 4)
            bits_4_to_fp6_4_packed<uint16_t, BF16_SPEC>(bf16_ptr + i, fp6_ptr + i / 4 * 3);

    } else {
        throw std::invalid_argument("Only FP32, FP16, and BF16 inputs are accepted.");
    }

    return fp6_tensor;
}

template <typename T, uint32_t FP_SPEC, int BLOCK_SIZE>
__global__ void bits_to_fp6_packed_kernel(const T *bits_ptr, uint8_t *fp6_ptr, int n) {
    // naive version
    // times 4 since each thread will handle 4 values
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (idx < n)
        bits_4_to_fp6_4_packed<T, FP_SPEC>(bits_ptr + idx, fp6_ptr + idx / 4 * 3);
    return;

    // more optimized version. coalesced memory write (speedup is minimal)
    // const int tid = threadIdx.x;
    // const int input_offset = (blockIdx.x * blockDim.x) * 4;
    // const int output_offset = (blockIdx.x * blockDim.x) * 3;

    // bits_ptr += input_offset;
    // fp6_ptr += output_offset;

    // __shared__ uint8_t shmem[BLOCK_SIZE * 3];

    // if (input_offset + tid * 4 < n) {
    //     uint8_t val0, val1, val2, val3;
    //     if (std::is_same_v<T, uint32_t>) {
    //         uint4 values = reinterpret_cast<const uint4 *>(bits_ptr)[tid * 4];
    //         val0 = bits_to_fp6<T, FP_SPEC>(values.x);
    //         val1 = bits_to_fp6<T, FP_SPEC>(values.y);
    //         val2 = bits_to_fp6<T, FP_SPEC>(values.z);
    //         val3 = bits_to_fp6<T, FP_SPEC>(values.w);
    //     } else if (std::is_same_v<T, uint16_t>) {
    //         ushort4 values = reinterpret_cast<const ushort4 *>(bits_ptr)[tid * 4];
    //         val0 = bits_to_fp6<T, FP_SPEC>(values.x);
    //         val1 = bits_to_fp6<T, FP_SPEC>(values.y);
    //         val2 = bits_to_fp6<T, FP_SPEC>(values.z);
    //         val3 = bits_to_fp6<T, FP_SPEC>(values.w);
    //     } else {
    //         val0 = bits_to_fp6<T, FP_SPEC>(bits_ptr[tid * 4]);
    //         val1 = bits_to_fp6<T, FP_SPEC>(bits_ptr[tid * 4 + 1]);
    //         val2 = bits_to_fp6<T, FP_SPEC>(bits_ptr[tid * 4 + 2]);
    //         val3 = bits_to_fp6<T, FP_SPEC>(bits_ptr[tid * 4 + 3]);
    //     }
    //     shmem[tid * 3]     = (val0 << 2) | (val1 >> 4);  // 0000 0011
    //     shmem[tid * 3 + 1] = (val1 << 4) | (val2 >> 2);  // 1111 2222
    //     shmem[tid * 3 + 2] = (val2 << 6) | (val3);       // 2233 3333
    // }
    // __syncthreads();

    // // TODO: write in larger word size
    // for (int i = 0; i < 3; i++) {
    //     if (output_offset + BLOCK_SIZE * i + tid < n / 4 * 3) {
    //         fp6_ptr[BLOCK_SIZE * i + tid] = shmem[BLOCK_SIZE * i + tid];
    //     }
    // }
}

at::Tensor to_fp6_packed_cuda(at::Tensor fp_tensor) {
    TORCH_CHECK(fp_tensor.is_contiguous());
    TORCH_CHECK(fp_tensor.is_cuda());
    TORCH_CHECK(fp_tensor.ndimension() == 2);

    int M = fp_tensor.size(0);
    int N = fp_tensor.size(1);
    TORCH_CHECK(N % 4 == 0, "Last dimension must be a multiple of 4, receives ", N);

    at::TensorOptions options = at::TensorOptions().dtype(torch::kUInt8).device(fp_tensor.device());
    at::Tensor fp6_tensor = at::empty({M, N * 3 / 4}, options);
    uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();

    int n = fp_tensor.numel();
    auto dtype = fp_tensor.dtype();

    // times 4 since each thread will handle 4 values
    constexpr int block_size = 256;
    int grid_size = (n + (block_size * 4) - 1) / (block_size * 4);

    if (dtype == torch::kFloat32) {
        const uint32_t *fp32_ptr = reinterpret_cast<uint32_t *>(fp_tensor.data_ptr<float>());
        bits_to_fp6_packed_kernel<uint32_t, FP32_SPEC, block_size><<<grid_size, block_size>>>(fp32_ptr, fp6_ptr, n);

    } else if (dtype == torch::kFloat16) {
        const uint16_t *fp16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::Half>());
        bits_to_fp6_packed_kernel<uint16_t, FP16_SPEC, block_size><<<grid_size, block_size>>>(fp16_ptr, fp6_ptr, n);

    } else if (dtype == torch::kBFloat16) {
        const uint16_t *bf16_ptr = reinterpret_cast<uint16_t *>(fp_tensor.data_ptr<at::BFloat16>());
        bits_to_fp6_packed_kernel<uint16_t, BF16_SPEC, block_size><<<grid_size, block_size>>>(bf16_ptr, fp6_ptr, n);

    } else {
        throw std::invalid_argument("Only FP32, FP16, and BF16 inputs are accepted.");
    }

    return fp6_tensor;
}

__global__ void fp6_unpacked_to_fp32_kernel(const uint8_t *fp6_ptr, float *fp32_ptr, int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        fp32_ptr[idx] = fp6_to_fp32(fp6_ptr[idx]);
}

at::Tensor fp6_unpacked_to_fp32(at::Tensor fp6_tensor) {
    TORCH_CHECK(fp6_tensor.dtype() == torch::kUInt8);
    TORCH_CHECK(fp6_tensor.is_contiguous());
    TORCH_CHECK(fp6_tensor.is_cpu() || fp6_tensor.is_cuda());

    at::TensorOptions options = at::TensorOptions().dtype(torch::kFloat32).device(fp6_tensor.device());
    at::Tensor fp32_tensor = at::empty(fp6_tensor.sizes(), options);

    const uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();
    float *fp32_ptr = fp32_tensor.data_ptr<float>();
    int n = fp6_tensor.numel();

    if (fp6_tensor.is_cpu()) {
        #pragma omp parallel for
        for (int i = 0; i < n; i++)
            fp32_ptr[i] = fp6_to_fp32(fp6_ptr[i]);
    } else {
        constexpr int block_size = 256;
        int grid_size = (n + block_size * 4 - 1) / (block_size * 4);
        fp6_unpacked_to_fp32_kernel<<<grid_size, block_size>>>(fp6_ptr, fp32_ptr, n);
    }

    return fp32_tensor;
}

at::Tensor fp6_packed_to_fp32(at::Tensor fp6_tensor) {
    TORCH_CHECK(fp6_tensor.dtype() == torch::kUInt8);
    TORCH_CHECK(fp6_tensor.is_contiguous());
    TORCH_CHECK(fp6_tensor.is_cpu() || fp6_tensor.is_cuda());
    TORCH_CHECK(fp6_tensor.ndimension() == 2);

    int M = fp6_tensor.size(0);
    int N = fp6_tensor.size(1);
    TORCH_CHECK(N % 3 == 0, "Last dimension must be a multiple of 3, receives ", N);

    at::TensorOptions options = at::TensorOptions().dtype(torch::kFloat32).device(fp6_tensor.device());
    at::Tensor fp32_tensor = at::empty({M, N / 3 * 4}, options);

    const uint8_t *fp6_ptr = fp6_tensor.data_ptr<uint8_t>();
    float *fp32_ptr = fp32_tensor.data_ptr<float>();
    int n = fp6_tensor.numel();

    if (fp6_tensor.is_cpu()) {
        #pragma omp parallel for
        for (int i = 0; i < n; i += 3)
            fp6_4_packed_to_fp32_4(fp6_ptr + i, fp32_ptr + i / 3 * 4);
    } else {
        constexpr int block_size = 256;
        int grid_size = (n + block_size * 3 - 1) / (block_size * 3);
        fp6_packed_to_fp32_kernel<<<grid_size, block_size>>>(fp6_ptr, fp32_ptr, n);
    }

    return fp32_tensor;
}

TORCH_LIBRARY_IMPL(torchao, CPU, m) {
  m.impl("torchao::to_fp6_unpacked", &to_fp6_unpacked_cpu);
  m.impl("torchao::to_fp6_packed", &to_fp6_packed_cpu);
  m.impl("torchao::fp6_unpacked_to_fp32", &fp6_unpacked_to_fp32);
  m.impl("torchao::fp6_packed_to_fp32", &fp6_packed_to_fp32);
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::to_fp6_unpacked", &to_fp6_unpacked_cuda);
  m.impl("torchao::to_fp6_packed", &to_fp6_packed_cuda);
  m.impl("torchao::fp6_unpacked_to_fp32", &fp6_unpacked_to_fp32);
  m.impl("torchao::fp6_packed_to_fp32", &fp6_packed_to_fp32);
}

}
