#include "hip/hip_runtime.h"
//    Copyright 2024 FP6-LLM authors
//
//    Licensed under the Apache License, Version 2.0 (the "License");
//    you may not use this file except in compliance with the License.
//    You may obtain a copy of the License at
//
//        http://www.apache.org/licenses/LICENSE-2.0
//
//    Unless required by applicable law or agreed to in writing, software
//    distributed under the License is distributed on an "AS IS" BASIS,
//    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//    See the License for the specific language governing permissions and
//    limitations under the License.
// 
// This file is adapted from https://github.com/usyd-fsalab/fp6_llm/blob/ce76774bcfc26b325c1b558abcf1935026d9abbc/fp6_llm/csrc/fp6_linear.cu

#include "kernel_matmul.cuh"
#include "kernel_reduction.cuh"

#include <stdio.h>
#include <assert.h>

template<typename TilingConfig, typename OutputDataType>
static void Kernel_Ex(hipStream_t    stream,
                      const uint4     *Weight,
                      const half      *Scales,
                      const half      *B,
                      OutputDataType  *C,
                      const size_t    M_Global,
                      const size_t    N_Global,
                      const size_t    K_Global, 
                      int             Split_K) 
{
    #ifdef DEBUG_MODE
        printf("\n");
        printf("Launcher.cu->Kernel_Ex():\n");
        printf("M: %d, N: %d, K: %d, SplitK: %d\n", M_Global, N_Global, K_Global, Split_K);
        printf("TILE_M: %d, TILE_K: %d, TILE_N: %d\n", TilingConfig::TILE_M, TilingConfig::TILE_K, TilingConfig::TILE_N);
    #endif
    static size_t SHMEM_SZ = max(TilingConfig::SMEM_SIZE_B_TILE+SMEM_SIZE_A1_TILE+SMEM_SIZE_A2_TILE, TilingConfig::SMEM_SIZE_C_TILE);
    hipFuncSetAttribute(reinterpret_cast<const void*>(QUANT_GEMM_Kernel<TilingConfig), OutputDataType>, hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ);
    size_t  dimN = (N_Global-1) / TilingConfig::TILE_N + 1;
    size_t  dimM = M_Global * Split_K / TilingConfig::TILE_M;
    dim3    GridDim(dimN, dimM, 1);
    dim3    BlockDim(WARP_SIZE * TilingConfig::BLOCK_WARPS, 1, 1);
    //
    #ifdef DEBUG_MODE
        printf("GridDim.x: %d, GridDim.y: %d, GridDim.z: %d, BlockDim.x: %d, BlockDim.y: %d, BlockDim.z: %d SHMEM_SZ: %d\n",
                GridDim.x, GridDim.y, GridDim.z, BlockDim.x, BlockDim.y, BlockDim.z, SHMEM_SZ);
        printf("\n");
    #endif
    QUANT_GEMM_Kernel<TilingConfig, OutputDataType><<<GridDim, BlockDim, SHMEM_SZ, stream>>>
                    (Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);
}

/*
 *
 */
hipError_t fp6_linear_kernel(hipStream_t    stream,
                              const uint4     *Weight,
                              const half      *Scales,
                              const half      *B,
                              half            *C,
                              const size_t    M_Global,
                              const size_t    N_Global,
                              const size_t    K_Global, 
                              float           *Reduction_Workspace,  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
                              int             Split_K)
{
    assert(M_Global % 256 == 0);
    assert(K_Global % 64 == 0);
    assert(N_Global>0);

    // Work around to support more N shapes:
    size_t N_PowerOf2;
    if(N_Global>0 &&  N_Global<=8)      N_PowerOf2 = 8;
    if(N_Global>8 &&  N_Global<=16)     N_PowerOf2 = 16;
    if(N_Global>16 && N_Global<=32)     N_PowerOf2 = 32;
    if(N_Global>32 && N_Global<=64)     N_PowerOf2 = 64;
    if(N_Global>64 && N_Global<=128)    N_PowerOf2 = 128;
    if(N_Global>128)                    N_PowerOf2 = ((N_Global-1)/128+1) * 128;

    if (Split_K == 1) {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, half>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
        }
    }
    else {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, float>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
        }
        // Reduction for SplitK
        dim3 GridDim((M_Global * N_Global) / REDUCTION_ELEMENT_PER_THREADBLOCK, 1, 1);
        dim3 BlockDim(WARP_SIZE, 1, 1);
        SplitK_Reduction<<<GridDim, BlockDim, 0, stream>>>(C, Reduction_Workspace, M_Global, N_Global, Split_K);
    }
    return hipGetLastError();
}


#include <torch/extension.h>
#include <ATen/ATen.h>
#include <torch/library.h>

namespace torchao {
/*
Computes FP6-FP16 GEMM (PyTorch interface).

[Mathmatical Formula]
Standard definition of linear layer:    Out = In * trans(W), where In, Out, and W are stored in row-major.
After Equivalent transformation    :    trans(Out) = W * trans(In). Note that we do not perform "transpose" during runtime, we instead interpret the In/Out as column-major matrices when calling our CUDA kernel.

[Inputs]
  _in_feats:  tensor of shape [B, IC];                  // half 
  _weights:   int tensor of shape [OC, IC // 16 * 3];   // 3 INT32 words contains 16 FP6 weights.
  _scales:    tensor of shape [OC];                     // half
  splitK:     spliting the MatMul problem along K dimension for higher GPU utilization, default 1.
[Outputs]
  _out_feats: tensor of shape [B, OC];                  // half
*/
torch::Tensor fp6_linear_forward_cuda(torch::Tensor _in_feats,
                                      torch::Tensor _weights,
                                      torch::Tensor _scales,
                                      int64_t       splitK=1)
{
    int num_in_feats      = _in_feats.size(0);
    int num_in_channels   = _in_feats.size(1);
    int num_out_channels  = _weights.size(0);
    TORCH_CHECK(num_in_channels%64 == 0, "Expected in_features to be a multiple of 64, but received ", num_in_channels);
    TORCH_CHECK((num_in_channels/16*3) == _weights.size(1));    // Making sure the K dimension is matched.
    //
    int M = num_out_channels;
    int K = num_in_channels;
    int N = num_in_feats;
    // Input Tensors
    auto weight = reinterpret_cast<const uint4*>(_weights.data_ptr<int>());  // weights is [OC, IC] but in FP6.
    auto in_feats = reinterpret_cast<const half*>(_in_feats.data_ptr<at::Half>());
    auto scales   = reinterpret_cast<const half*>(_scales.data_ptr<at::Half>());
    // Output Tensors
    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty({num_in_feats, num_out_channels}, options);
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());

    options = torch::TensorOptions().dtype(torch::kFloat32).device(_in_feats.device());
    at::Tensor _workspace = torch::empty({splitK, num_in_feats, num_out_channels}, options);
    auto Reduction_Workspace = reinterpret_cast<float*>(_workspace.data_ptr<float>());  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
      
    fp6_linear_kernel(0, // Using default stream here.
                      weight,
                      scales,
                      in_feats,
                      out_feats,
                      M,
                      N,
                      K, 
                      Reduction_Workspace,  
                      splitK);

    return _out_feats;
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::fp16act_fp6weight_linear", &fp6_linear_forward_cuda);
}

} // namespace torchao
